#include "hip/hip_runtime.h"
//comment
#include "stdio.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "opencv4/opencv2/opencv.hpp"

#define BUF 616*800


extern "C" int test(int value);
extern "C" uchar getFrameCam(uchar *Frame,uchar *Frame_2,int buffer);



__global__ void add(int a, int b, int *c){
    *c = a + b;
}

__global__ void cudatxt(uchar *frame, uchar *frame_2, uchar *imagen){
    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if(j < BUF){
        //imagen[j] = pow(frame[j]-frame_2[j],2);
        imagen[j] = frame[j]*frame_2[j];
    }
}



int test(int value){
    printf("Hello %d \n", value);
    int c;
    int *dev_c;
    hipMalloc((void**)&dev_c,sizeof(int));
    add<<<1,1>>>(10,23,dev_c);
    hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);
    printf("2+7 = %d\n",c);
    hipFree(dev_c);
    return value;
}

uchar getFrameCam(uchar *Frame,uchar *Frame_2 ,int buffer){
    cv::Mat mat;
    uchar * dev_frame;
    uchar * dev_frame_2;
    uchar * dev_img;
    uchar imagen[616][800];
    uchar imbuff[buffer];
    hipMalloc((void**)&dev_frame, buffer*sizeof(uchar));
    hipMalloc((void**)&dev_frame_2, buffer*sizeof(uchar));
    hipMalloc((void**)&dev_img, buffer*sizeof(uchar));
    hipMemcpy(dev_frame,Frame,buffer*sizeof(uchar),hipMemcpyHostToDevice);
    hipMemcpy(dev_frame_2,Frame_2,buffer*sizeof(uchar),hipMemcpyHostToDevice);
    hipMemcpy(dev_img,imagen,buffer*sizeof(uchar),hipMemcpyHostToDevice);
    cudatxt<<<800,616>>>(dev_frame, dev_frame_2, dev_img);
    hipMemcpy(imagen,dev_img,buffer*sizeof(uchar),hipMemcpyDeviceToHost);
    mat = cv::Mat(616, 800, CV_8UC1, imagen);
    cv::imwrite("/home/nvxv/Pictures/out/sumaImagenes.bmp",mat);
    hipFree(dev_frame);
    hipFree(dev_frame_2);
    hipFree(dev_img);
    for(int i = 0; i<616; i++){
        for(int j = 0; j<800; j++){
            imbuff[j+(i*j)] = imagen[i][j];
        }
    }
    return imbuff[buffer];
}
